
#include <hip/hip_runtime.h>
﻿/*
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#define CHANNELS 3

__global__
void colorToGreyscaleConversion(unsigned char* Pout, unsigned
	char* Pin, int width, int height) {
	
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	if (Col < width && Row < height) {
		// get 1D coordinate for the grayscale image
		int greyOffset = Row * width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns than the grayscale image
		int rgbOffset = greyOffset * CHANNELS;
		unsigned char r = Pin[rgbOffset]; // red value for pixel
		unsigned char g = Pin[rgbOffset + 1]; // green value for pixel
		unsigned char b = Pin[rgbOffset + 2]; // blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		Pout[rgbOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
		Pout[rgbOffset+1] = 0.21f * r + 0.71f * g + 0.07f * b;
		Pout[rgbOffset+2] = 0.21f * r + 0.71f * g + 0.07f * b;
		

	}
}
int main(int arc,char ** argv) {
	int width, height, rgb;
	unsigned char* Pin = stbi_load(argv[1], &width, &height, &rgb, 3);
	//unsigned char* Pout = stbi_load(argv[1], &width, &height, &rgb, 3);
	unsigned char* ptrImageData = NULL;
	unsigned char* ptrImageDataOut = NULL;
	cudaMalloc(&ptrImageData, width * height * CHANNELS);
	cudaMemcpy(ptrImageData, Pin, width * height * CHANNELS, cudaMemcpyHostToDevice);
	cudaMalloc(&ptrImageDataOut, width * height * CHANNELS);
	cudaMemcpy(ptrImageDataOut, Pin, width * height * CHANNELS, cudaMemcpyHostToDevice);
	
	colorToGreyscaleConversion<<<dim3((width / 32), (height / 32)), dim3(32, 32) >>>(ptrImageData, ptrImageDataOut, width, height);

	cudaMemcpy(Pin, ptrImageData, width * height * CHANNELS, cudaMemcpyDeviceToHost);
	std::string NewImageFile = argv[1];
	NewImageFile = NewImageFile.substr(0, NewImageFile.find_last_of('.')) + "toGray.png";
	stbi_write_png(NewImageFile.c_str(), width, height, 3, Pin, 3 * width);
	cudaFree(ptrImageData);
	cudaFree(ptrImageDataOut);
	stbi_image_free(Pin);

}*/